#include "hip/hip_runtime.h"
#include <torch/types.h>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include "pixelUtils.cuh"
#include "arrayUtils.cuh"
#include "debugUtils.cuh"

__host__ __device__ float gaus(float x, float std){
    return expf(- (x * x) / (2.0f * std * std)) / (sqrtf(2.0f * M_PI) * std);
}

static __global__ void get_gaussian_kernel(
    float* kernel,
    float std, int size
) {
    int idx = threadIdx.x;
    int x = idx - size;
    extern __shared__ float sharedKernel[];

    float value = gaus(x, std);
    sharedKernel[idx] = value;
    __syncthreads();

    float sum = 0;
    for(int i = 0; i < blockDim.x; i++)
        sum += sharedKernel[i];

    kernel[idx] = value / sum;
}

template <typename scalar_t>
static __global__ void semi_conv_gray_kernel(
    scalar_t* result,
    const scalar_t* gray,
    const float* semi_kernel,
    int size, int pad
) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int h = gridDim.x;
    int w = gridDim.y - 2 * (pad - size);
    extern __shared__ float shared_buffer[];

    int t = threadIdx.x;
    int im_x = x-pad+t;
    int im_y = y+size-pad;
    scalar_t p = (im_x < 0 || im_x >= w) ? 0 : get_value(gray, im_x, im_y, h, w);

    float weight = semi_kernel[t];
    shared_buffer[t] = p * weight;
    __syncthreads();

    if(t != 0)
        return;

    scalar_t r = 0.0;
    for(int i = 0; i<2*size+1; i++){
        r += shared_buffer[i];
    }
    set_value(result, r, y, x); // transpose
}

template <typename scalar_t>
static __global__ void semi_bilateral_conv_gray_kernel(
    scalar_t* result,
    const scalar_t* gray,
    const float* semi_kernel,
    float std, int size, int pad
) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int h = gridDim.x;
    int e = size-pad;
    int w = gridDim.y + 2 * e;
    extern __shared__ float shared_buffer[];

    int t = threadIdx.x;
    int im_x = x-pad+t;
    int im_y = y+size-pad;
    scalar_t center = get_value(gray, x+e, y+e, h, w);
    scalar_t p = (im_x < 0 || im_x >= w) ? 0 : get_value(gray, im_x, im_y, h, w);

    float weight1 = semi_kernel[t];
    float weight2 = gaus(center-p, std);
    shared_buffer[t]            = p * weight1 * weight2;
    shared_buffer[t + 2*size+1] = weight1 * weight2;
    __syncthreads();

    if(t != 0)
        return;

    scalar_t r = 0.0;
    scalar_t norm = 0.0;
    for(int i = 0; i<2*size+1; i++){
        r += shared_buffer[i];
        norm += shared_buffer[i + 2*size+1];
    }
    r /= norm;
    set_value(result, r, y, x); // transpose
}

template <typename scalar_t>
static __global__ void semi_conv_kernel(
    scalar_t* result,
    const scalar_t* image,
    const float* semi_kernel,
    int size, int pad
) {
    int x = blockIdx.x;
    int y = blockIdx.y;

    pixel<scalar_t> r;
    for(int i = -size; i<size+1; i++){
        float w = semi_kernel[i+size];
        pixel<scalar_t> p = get_pixel(image, x, y);
    }
    // TODO: to be finished.
    get_pixel(image, x, y);
}

template <typename scalar_t>
static __global__ void median_kernel(
    scalar_t* result,
    const scalar_t* gray,
    int size, int pad, bool pseudo
) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int h = gridDim.x - 2 * (pad - size);
    int w = gridDim.y - 2 * (pad - size);
    int n = size*2+1;
    int len = n*n;
    int m = (len+1)/2;

    scalar_t* array = (scalar_t*)malloc(len * sizeof(scalar_t));
    for(int i = -size; i<size+1; i++){
        for(int j = -size; j<size+1; j++){
            int im_x = x+size-pad+i;
            int im_y = y+size-pad+j;

            bool out = (im_x < 0 || im_x >= w) || (im_y < 0 || im_y >= h);
            array[(i+size)*n + j+size] = out ? 0.0 : get_value(gray, im_x, im_y, h, w);
        }
    }

    scalar_t median = 0.0;
    if(pseudo){
        scalar_t* temp  = (scalar_t*)malloc((len-m+1) * sizeof(scalar_t));
        scalar_t minmax_v = arr::minmax(array, temp, len);
        scalar_t maxmin_v = arr::maxmin(array, temp, len);
        median = (minmax_v + maxmin_v)/2.0;
        free(temp);
    }else{
        median = arr::median(array, len);
    }

    set_value(result, median, x, y);
    free(array);
}

// C++ API

void separable_conv_op(
    torch::Tensor& result,
    const torch::Tensor& image,
    const float* kernel,
    int size, int pad
) {
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);

    int b = image.size(0);
    int c = image.size(1);
    int h = image.size(2);
    int w = image.size(3);
    int e = pad - size;
    int l = 2*size+1;

    torch::Tensor temp = torch::empty({b, c, w+2*e, h}).to(image.device()); // transpose
    dim3 grid_size1(h,     w+2*e, b);
    dim3 grid_size2(h+2*e, w+2*e, b);

    if(c == 3){
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "semi_conv_kernel", [&] {
            semi_conv_kernel<scalar_t><<<grid_size1, l, l*sizeof(float), stream>>>(
                temp.data_ptr<scalar_t>(),
                image.data_ptr<scalar_t>(),
                kernel,
                size, pad
            );
        });
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "semi_conv_kernel", [&] {
            semi_conv_kernel<scalar_t><<<grid_size2, l, l*sizeof(float), stream>>>(
                result.data_ptr<scalar_t>(),
                temp.data_ptr<scalar_t>(),
                kernel,
                size, pad
            );
        });
    }else if (c == 1){
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "semi_conv_gray_kernel", [&] {
            semi_conv_gray_kernel<scalar_t><<<grid_size1, l, l*sizeof(float), stream>>>(
                temp.data_ptr<scalar_t>(),
                image.data_ptr<scalar_t>(),
                kernel,
                size, pad
            );
        });
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "semi_conv_gray_kernel", [&] {
            semi_conv_gray_kernel<scalar_t><<<grid_size2, l, l*sizeof(float), stream>>>(
                result.data_ptr<scalar_t>(),
                temp.data_ptr<scalar_t>(),
                kernel,
                size, pad
            );
        });
    }
}

void uniform_conv_op(
    torch::Tensor& result,
    const torch::Tensor& image,
    int size, int pad
) {
    float* kernel = make_array<float>(2*size+1, 1.0/(2.0*size+1.0));
    separable_conv_op(result, image, kernel, size, pad);
    hipFree(kernel);
}

void gaussian_conv_op(
    torch::Tensor& result,
    const torch::Tensor& image,
    float std, int size, int pad
) {
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);

    float* kernel = make_array<float>(2*size+1, 0);
    get_gaussian_kernel<<<1, 2*size+1, (2*size+1)*sizeof(float), stream>>>(kernel, std, size);
    separable_conv_op(result, image, kernel, size, pad);
    hipFree(kernel);
}

void median_filter_op(
    torch::Tensor& result,
    const torch::Tensor& image,
    int size, int pad, bool pseudo
) {
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);

    int b = image.size(0);
    int h = image.size(2);
    int w = image.size(3);
    int e = pad - size;
    int n = 2*size+1;

    if(pseudo){
        hipDeviceSetLimit(hipLimitMallocHeapSize, 2*n*n*sizeof(float));
    }else{
        hipDeviceSetLimit(hipLimitMallocHeapSize, 1*n*n*sizeof(float));
        hipDeviceSetLimit(hipLimitStackSize, n*n*1024+1024);
    }

    dim3 grid_size(h+2*e, w+2*e, b);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "median_kernel", [&] {
        median_kernel<scalar_t><<<grid_size, 1, 0, stream>>>(
            result.data_ptr<scalar_t>(),
            image.data_ptr<scalar_t>(),
            size, pad, pseudo
        );
    });
}

#define PI 3.14159265358979323846
void bilateral_filter_op(
    torch::Tensor& result,
    const torch::Tensor& image,
    float std_k, float std_i, int size, int pad
){
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);

    int b = image.size(0);
    int c = image.size(1);
    int h = image.size(2);
    int w = image.size(3);
    int e = pad - size;
    int l = 2*size+1;

    float* kernel = make_array<float>(2*size+1, 0);
    get_gaussian_kernel<<<1, 2*size+1, (2*size+1)*sizeof(float), stream>>>(kernel, std_k, size);

    torch::Tensor temp = torch::empty({b, c, w+2*e, h}).to(image.device()); // transpose
    dim3 grid_size1(h,     w+2*e, b);
    dim3 grid_size2(h+2*e, w+2*e, b);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "semi_bilateral_conv_gray_kernel", [&] {
        semi_bilateral_conv_gray_kernel<scalar_t><<<grid_size1, l, 2*l*sizeof(float), stream>>>(
            temp.data_ptr<scalar_t>(),
            image.data_ptr<scalar_t>(),
            kernel, std_i,
            size, pad
        );
    });
    float gau_2 = 0.0;
    for(int i = -size; i<size+1; i++){
        float gau = gaus(i, std_k);
        gau_2 += gau * gau;
    }
    float fact = gau_2 / (2*PI*std_i*std_i + 4*PI*std_k*std_k);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "semi_bilateral_conv_gray_kernel", [&] {
        semi_bilateral_conv_gray_kernel<scalar_t><<<grid_size2, l, 2*l*sizeof(float), stream>>>(
            result.data_ptr<scalar_t>(),
            temp.data_ptr<scalar_t>(),
            kernel, std_i,// / sqrtf(fact),
            size, pad
        );
    });
}