#include "hip/hip_runtime.h"
#include <torch/types.h>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include "pixelUtils.cuh"
#include "arrayUtils.cuh"

__host__ __device__ float gaus(float x, float std){
    return expf(- (x * x) / (2.0f * std * std)) / (sqrtf(2.0f * M_PI) * std);
}

static __global__ void get_gaussian_kernel(
    float* kernel,
    float std, int size
) {
    int idx = threadIdx.x;
    int x = idx - size;
    extern __shared__ float sharedKernel[];

    float value = gaus(x, std);
    sharedKernel[idx] = value;
    __syncthreads();

    float sum = 0;
    for(int i = 0; i < blockDim.x; i++)
        sum += sharedKernel[i];

    kernel[idx] = value / sum;
}

template <typename scalar_t>
static __global__ void semi_conv_gray_kernel(
    scalar_t* result,
    const scalar_t* gray,
    const float* semi_kernel,
    int size, int pad
) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int h = gridDim.x;
    int w = gridDim.y - 2 * (pad - size);

    scalar_t r = 0.0;
    for(int i = -size; i<size+1; i++){
        int im_x = x+size-pad+i;
        int im_y = y+size-pad;

        float weight = semi_kernel[i+size];
        scalar_t p = (im_x < 0 || im_x >= w) ? 0 : get_value(gray, im_x, im_y, h, w);
        r += p * weight;
    }

    set_value(result, r, y, x); // transpose
}

template <typename scalar_t>
static __global__ void semi_bilateral_conv_gray_kernel(
    scalar_t* result,
    const scalar_t* gray,
    const float* semi_kernel,
    float std, int size, int pad
) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int h = gridDim.x;
    int e = size-pad;
    int w = gridDim.y + 2 * e;

    scalar_t center = get_value(gray, x+e, y+e, h, w);
    scalar_t r = 0.0;
    scalar_t norm = 0.0;
    for(int i = -size; i<size+1; i++){
        int im_x = x+e+i;
        int im_y = y+e;

        scalar_t p = (im_x < 0 || im_x >= w) ? 0 : get_value(gray, im_x, im_y, h, w);
        float weight1 = semi_kernel[i+size];
        float weight2 = gaus(center-p, std);
        r += p * weight1 * weight2;
        norm += weight1 * weight2;
    }
    r /= norm;
    set_value(result, r, y, x); // transpose
}

template <typename scalar_t>
static __global__ void semi_conv_kernel(
    scalar_t* result,
    const scalar_t* image,
    const float* semi_kernel,
    int size, int pad
) {
    int x = blockIdx.x;
    int y = blockIdx.y;

    pixel<scalar_t> r;
    for(int i = -size; i<size+1; i++){
        float w = semi_kernel[i+size];
        pixel<scalar_t> p = get_pixel(image, x, y);
    }
    // TODO: to be finished.
    get_pixel(image, x, y);
}

template <typename scalar_t>
static __global__ void median_kernel(
    scalar_t* result,
    const scalar_t* gray,
    int size, int pad, bool pseudo
) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int h = gridDim.x - 2 * (pad - size);
    int w = gridDim.y - 2 * (pad - size);
    int n = size*2+1;
    int len = n*n;
    int m = (len+1)/2;

    scalar_t* array = (scalar_t*)malloc(len * sizeof(scalar_t));
    for(int i = -size; i<size+1; i++){
        for(int j = -size; j<size+1; j++){
            int im_x = x+size-pad+i;
            int im_y = y+size-pad+j;

            bool out = (im_x < 0 || im_x >= w) || (im_y < 0 || im_y >= h);
            array[(i+size)*n + j+size] = out ? 0.0 : get_value(gray, im_x, im_y, h, w);
        }
    }

    scalar_t median = 0.0;
    if(pseudo){
        scalar_t* temp  = (scalar_t*)malloc((len-m+1) * sizeof(scalar_t));
        scalar_t minmax_v = arr::minmax(array, temp, len);
        scalar_t maxmin_v = arr::maxmin(array, temp, len);
        median = (minmax_v + maxmin_v)/2.0;
        free(temp);
    }else{
        median = arr::median(array, len);
    }

    set_value(result, median, x, y);
    free(array);
}

// C++ API

void separable_conv_op(
    torch::Tensor& result,
    const torch::Tensor& image,
    const float* kernel,
    int size, int pad
) {
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);

    int b = image.size(0);
    int c = image.size(1);
    int h = image.size(2);
    int w = image.size(3);
    int e = pad - size;

    torch::Tensor temp = torch::empty({b, c, w+2*e, h}).to(image.device()); // transpose
    dim3 grid_size1(h,     w+2*e, 1);
    dim3 grid_size2(h+2*e, w+2*e, 1);

    if(c == 3){
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "semi_conv_kernel", [&] {
            semi_conv_kernel<scalar_t><<<grid_size1, b, 0, stream>>>(
                temp.data_ptr<scalar_t>(),
                image.data_ptr<scalar_t>(),
                kernel,
                size, pad
            );
        });
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "semi_conv_kernel", [&] {
            semi_conv_kernel<scalar_t><<<grid_size2, b, 0, stream>>>(
                result.data_ptr<scalar_t>(),
                temp.data_ptr<scalar_t>(),
                kernel,
                size, pad
            );
        });
    }else if (c == 1){
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "semi_conv_gray_kernel", [&] {
            semi_conv_gray_kernel<scalar_t><<<grid_size1, b, 0, stream>>>(
                temp.data_ptr<scalar_t>(),
                image.data_ptr<scalar_t>(),
                kernel,
                size, pad
            );
        });
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "semi_conv_gray_kernel", [&] {
            semi_conv_gray_kernel<scalar_t><<<grid_size2, b, 0, stream>>>(
                result.data_ptr<scalar_t>(),
                temp.data_ptr<scalar_t>(),
                kernel,
                size, pad
            );
        });
    }
}

void uniform_conv_op(
    torch::Tensor& result,
    const torch::Tensor& image,
    int size, int pad
) {
    float* kernel = make_array<float>(2*size+1, 1.0/(2.0*size+1.0));
    separable_conv_op(result, image, kernel, size, pad);
    hipFree(kernel);
}

void gaussian_conv_op(
    torch::Tensor& result,
    const torch::Tensor& image,
    float std, int size, int pad
) {
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);

    float* kernel = make_array<float>(2*size+1, 0);
    get_gaussian_kernel<<<1, 2*size+1, (2*size+1)*sizeof(float), stream>>>(kernel, std, size);
    separable_conv_op(result, image, kernel, size, pad);
    hipFree(kernel);
}

void median_filter_op(
    torch::Tensor& result,
    const torch::Tensor& image,
    int size, int pad, bool pseudo
) {
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);

    int b = image.size(0);
    int h = image.size(2);
    int w = image.size(3);
    int e = pad - size;

    dim3 grid_size(h+2*e, w+2*e, 1);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "median_kernel", [&] {
        median_kernel<scalar_t><<<grid_size, b, 0, stream>>>(
            result.data_ptr<scalar_t>(),
            image.data_ptr<scalar_t>(),
            size, pad, pseudo
        );
    });
}

void bilateral_filter_op(
    torch::Tensor& result,
    const torch::Tensor& image,
    float std_k, float std_i, int size, int pad
){
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);

    int b = image.size(0);
    int c = image.size(1);
    int h = image.size(2);
    int w = image.size(3);
    int e = pad - size;

    float* kernel = make_array<float>(2*size+1, 0);
    get_gaussian_kernel<<<1, 2*size+1, (2*size+1)*sizeof(float), stream>>>(kernel, std_k, size);

    torch::Tensor temp = torch::empty({b, c, w+2*e, h}).to(image.device()); // transpose
    dim3 grid_size1(h,     w+2*e, 1);
    dim3 grid_size2(h+2*e, w+2*e, 1);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "semi_bilateral_conv_gray_kernel", [&] {
        semi_bilateral_conv_gray_kernel<scalar_t><<<grid_size1, b, 0, stream>>>(
            temp.data_ptr<scalar_t>(),
            image.data_ptr<scalar_t>(),
            kernel, std_i,
            size, pad
        );
    });
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "semi_bilateral_conv_gray_kernel", [&] {
        semi_bilateral_conv_gray_kernel<scalar_t><<<grid_size2, b, 0, stream>>>(
            result.data_ptr<scalar_t>(),
            temp.data_ptr<scalar_t>(),
            kernel, std_i,
            size, pad
        );
    });
}