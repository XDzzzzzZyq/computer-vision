#include "hip/hip_runtime.h"
#include <torch/types.h>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "pixelUtils.cuh"
#include "arrayUtils.cuh"

template <typename scalar_t>
static __global__ void minmax_kernel(
    int* batched_min, int* batched_max,
    const scalar_t* image
) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int b = threadIdx.x;

    scalar_t gray = get_value(image, x, y);
    atomicMin(&batched_min[b], int(gray));
    atomicMax(&batched_max[b], int(gray));
}

template <typename scalar_t>
static __global__ void minmax_scale_kernel(
    scalar_t* result,
    const scalar_t* image,
    const int* batched_min, const int* batched_max
) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int b = threadIdx.x;

    scalar_t gray = get_value(image, x, y);
    scalar_t scaled = (gray - batched_min[b])/(batched_max[b] - batched_min[b]) * 255.;
    set_value(result, scaled, x, y);
}

// C++ API

void minmax_scale_op(
    torch::Tensor& result,
    const torch::Tensor& image
) {
    int curDevice = -1;
    hipGetDevice(&curDevice);
    hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);

    int b = image.size(0);
    int h = image.size(2);
    int w = image.size(3);
    dim3 grid_size(h, w, 1);

    int* batched_min = make_array(b, INT_MAX);
    int* batched_max = make_array(b, INT_MIN);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "minmax_kernel", [&] {
        minmax_kernel<scalar_t><<<grid_size, b, 0, stream>>>(
            batched_min, batched_max,
            image.data_ptr<scalar_t>()
        );
    });
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(image.scalar_type(), "minmax_scale_kernel", [&] {
        minmax_scale_kernel<scalar_t><<<grid_size, b, 0, stream>>>(
            result.data_ptr<scalar_t>(),
            image.data_ptr<scalar_t>(),
            batched_min, batched_max
        );
    });

    hipFree(batched_min);
    hipFree(batched_max);
}